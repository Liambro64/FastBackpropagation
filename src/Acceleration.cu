#include "hip/hip_runtime.h"
#include "../Project.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" __global__ void outerProductGPU(double **to, double *a, double *b, int asize, int bsize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < asize)
		for (int j = 0; j < bsize; ++j)
			to[idx][j] = a[idx] * b[j];
}

extern "C" __global__ void vectorMatrixMultiplyGPU(ddd *to, ddd *vector, ddd *matrix, int vectorSize, int matrixRows, int numpitch)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < vectorSize)
	{
		matrix += idx; // Adjust pointer to the correct column
		ddd sum = 0.0;
		for (int i = 0; i < matrixRows; i++)
		{
			sum += vector[idx] * matrix[i * numpitch];
		}
		to[idx] = sum;
	}
}

extern "C" __global__ void weightedSumGPU(ddd *to, ddd *outsideValues, ddd *insideValues, int outsideSize, int insideSize, int numpitch)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < insideSize)
	{
		insideValues += idx * numpitch; // Adjust pointer to the correct row
		ddd sum = insideValues[0];		// Start with the bias
		for (int i = 0; i < outsideSize; i++)
		{
			sum += outsideValues[i] * insideValues[i + 1];
		}
		to[idx] = 1 / (1 + exp(-sum));
	}
}

extern "C" __global__ void weightedSumGPUInside(ddd *to, ddd *outsideValues, ddd **ptrs, int ptrIndex, int outsideSize, int insideSize, int numpitch)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	double *insideValues = ptrs[ptrIndex]; // Get the pointer to the inside values for this index
	if (idx < insideSize)
	{
		insideValues += idx * numpitch; // Adjust pointer to the correct row
		ddd sum = insideValues[0];		// Start with the bias
		for (int i = 0; i < outsideSize; i++)
		{
			sum += outsideValues[i] * insideValues[i + 1];
		}
		to[idx] = 1 / (1 + exp(-sum));
	}
}
// extern "C" __global__ void vectorSum(int *to, int **from, int size) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;

//     if (idx < size) {
//         int *current_from_array = from[idx];

//         // Ensure the pointer is not null before dereferencing
//         if (current_from_array != nullptr) {
//             int num_elements_to_sum = current_from_array[0];
//             int current_sum = 0;

//             // Sum elements from current_from_array[1] to current_from_array[num_elements_to_sum]
//             for (int j = 1; j <= num_elements_to_sum; j++) {
//                 current_sum += current_from_array[j];
//             }
//             to[idx] = current_sum;
//         }
//     }
// }

// namespace Wrapper{

// void VSWrapper(int *to, int **from, int size) {
//     // Define the number of threads and blocks
//     int threadsPerBlock = 256;
//     int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

//     // Launch the kernel
//     vectorSum<<<blocksPerGrid, threadsPerBlock>>>(to, from, size);
//     // Check for errors in kernel launch (optional)
//     hipError_t err = hipGetLastError();
//     if (err != hipSuccess) {
//         fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
//     }
// 	err = hipDeviceSynchronize(); // Wait for kernel to complete
//     if (err != hipSuccess) {
//         fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
//     }
// }
// }

// should be a wrapper functuon to run multiple weighted sums in parallel
extern "C" vec<ddd> weightedSumsWp(vec<ddd> outsideValues, vec<vec<ddd>> insideValues)
{
	double *h_insides = new double [insideValues.size() * insideValues[0].size()];
	for (size_t i = 0; i < insideValues.size(); ++i)
	{
		for (size_t j = 0; j < insideValues[i].size(); ++j)
		{
			h_insides[i * insideValues[0].size() + j] = insideValues[i][j];
		}
	}
	double *d_outsideValues;
	double *d_insideValues;
	double *returnVals;
	size_t pitch;
	// allocate and copy inside values to device
	if (hipMalloc(&d_outsideValues, outsideValues.size() * sizeof(double)) != hipSuccess)
		std::cout << "Failed to alloc 1" << std::endl;

	if (hipMemcpy(d_outsideValues, outsideValues.data(), outsideValues.size() * sizeof(double), hipMemcpyHostToDevice) != hipSuccess)
		std::cout << "Failed to copy 1" << std::endl;
	if (hipMallocPitch(&d_insideValues, &pitch, sizeof(double) * insideValues[0].size(), insideValues.size()) != hipSuccess)
		std::cout << "Failed to alloc 2 (pitch)" << std::endl;
	if (hipMemcpy2D(d_insideValues, pitch, h_insides, insideValues[0].size() * sizeof(ddd),
					 insideValues[0].size() * sizeof(ddd), insideValues.size(), hipMemcpyHostToDevice) != hipSuccess)
		std::cout << "Failed to copy 2" << std::endl;
	if (hipMalloc(&returnVals, insideValues.size() * sizeof(double)) != hipSuccess)
		std::cout << "Failed to alloc 3" << std::endl;
	int isize = insideValues.size();
	int osize = outsideValues.size();
	int threadsPerBlock = 256;
	int blocksPerGrid = (insideValues.size() + threadsPerBlock - 1) / threadsPerBlock;
	dim3 blockSize = dim3(threadsPerBlock, 1, 1);
	dim3 gridSize = dim3(blocksPerGrid, 1, 1);
	weightedSumGPU<<<gridSize, blockSize>>>((double *)returnVals, (double *)d_outsideValues, d_insideValues, osize, isize, pitch / sizeof(double));

	hipDeviceSynchronize();

	vec<ddd> result(insideValues.size());
	hipMemcpy(result.data(), returnVals, insideValues.size() * sizeof(ddd), hipMemcpyDeviceToHost);

	delete h_insides;
	return result;
}

extern "C" vec<vec<ddd>> FullRun(vec<ddd> input, vec<vec<vec<ddd>>> weights) {
	// input is a vector of doubles
	// weights is a vector of matrices, each matrix is a vector of vectors of doubles
	size_t size = 0;
	vec<size_t> sizes(weights.size());
	for (int i = 0; i < weights.size(); i++) {
		sizes[i] = weights[i].size() * weights[i][0].size();
		size += sizes[i] * sizeof(double);
	}
	double *h_flatweights = new double[size];
	int i = 0;
	for (int j = 0; j < weights.size(); j++) {
		for (int k = 0; k < weights[j].size(); k++) {
			for (int l = 0; l < weights[j][k].size(); l++) {
				h_flatweights[i++] = weights[j][k][l];
			}
		}
	}
	// this is an array (on the host) of pointers (on the device) to the weights
	double **d_weights = new double *[weights.size()];
	//this is for the array of pointers after its copied on the device
	double **d_weights_pointers;
	//the inputs on the device
	double *d_input;
	//the return value (on the device) but also reused as the input layer
	double *d_returnVals;

	size_t maxInputSize = MAX(max(sizes), input.size());

	size_t *pitches = new size_t[weights.size()];
	for (int i = 0; i < weights.size(); i++) {
		hipMallocPitch(&(d_weights[i]), &(pitches[i]), weights[i][0].size() * sizeof(double), weights[i].size());
		hipMemcpy2D(d_weights[i], pitches[i], h_flatweights + sumFor(sizes, i), weights[i][0].size() * sizeof(double),
				 weights[i][0].size() * sizeof(double), weights[i].size(), hipMemcpyHostToDevice);
	}
	hipMalloc(&(d_weights_pointers), weights.size() * sizeof(double *));
	hipMemcpy(d_weights_pointers, d_weights, weights.size() * sizeof(double *), hipMemcpyHostToDevice);
	hipMalloc(&d_input, maxInputSize * sizeof(double));
	hipMemcpy(d_input, input.data(), input.size() * sizeof(double), hipMemcpyHostToDevice);
	hipMalloc(&d_returnVals, maxInputSize * sizeof(double));
	int threadsPerBlock = 256;
	vec<vec<ddd>> returnVals(weights.size());
	for (int i = 0; i < weights.size(); i++) {
		returnVals[i].resize(weights[i].size());
		dim3 block = dim3(threadsPerBlock, 1, 1);
		dim3 grid = dim3(((i == 0 ? input.size() : weights[i - 1].size()) + threadsPerBlock - 1) / threadsPerBlock, 1, 1);
		weightedSumGPUInside<<<block, grid>>>(d_returnVals, d_input, d_weights_pointers, i, i == 0 ? input.size() : sizes[i - 1], weights[i].size(), pitches[i] / sizeof(double));
		hipDeviceSynchronize();
		hipMemcpy(returnVals[i].data(), d_returnVals, weights[i].size() * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(d_input, d_returnVals, weights[i].size() * sizeof(double), hipMemcpyDeviceToDevice);
	}

	return returnVals;
}
